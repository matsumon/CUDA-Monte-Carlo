#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"



// how many tries to discover the maximum performance:
#ifndef BLOCKSIZE
#define BLOCKSIZE	512
#endif

// how many tries to discover the maximum performance:
#ifndef NUMBLOCKS
#define NUMBLOCKS	(NUMTRIALS/BLOCKSIZE)
#endif

#define IN
#define OUT
// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	1024
#endif
// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL

// call this if you want to force your program to use
// a different random number sequence every time you run it:
void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

void
CudaCheckError( )
{
        hipError_t e = hipGetLastError( );
        if( e != hipSuccess )
        {
                fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
        }
}

float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}
// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
        return (M_PI/180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
        unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

        // randomize everything:
        float v   = dvs[gid];
        float thr = Radians( dths[gid] );
        float vx  = v * cos(thr);
        float vy  = v * sin(thr);
        float  g  =  dgs[gid];
        float  h  =  dhs[gid];
        float  d  =  dds[gid];

        int numHits = 0;
	// see if the ball doesn't even reach the cliff:`
	float t = -vy*2/GRAVITY; //edit
	float x = vx * t; //edit
	if( x <= g )
	{
		// if( DEBUG )	fprintf( stderr, "Ball doesn't even reach the cliff\n" );
	}
	else
	{
		// see if the ball hits the vertical cliff face:
		t = g/vx; //edit
		float y = vy * t + 0.5*GRAVITY*(t*t); //edit
		if( y <= h )
		{
			// if( DEBUG )	fprintf( stderr, "Ball hits the cliff face\n" );
		}
		else
		{
			// the ball hits the upper deck:
			// the time solution for this is a quadratic equation of the form:
			// at^2 + bt + c = 0.
			// where 'a' multiplies time^2
			//       'b' multiplies time
			//       'c' is a constant
			float a = 0.5*GRAVITY; //edit
			float b = vy; //edit
			// float c = 0; //edit
			float c = -h; //edit
			float disc = b*b - 4.f*a*c;	// quadratic formula discriminant

			// ball doesn't go as high as the upper deck:
			// this should "never happen" ... :-)
			if( disc < 0. )
			{
				// if( DEBUG )	fprintf( stderr, "Ball doesn't reach the upper deck.\n" );
				// exit( 1 );	// something is wrong...
			}
			else {
				// successfully hits the ground above the cliff:
				// get the intersection:
				disc = sqrtf( disc );
				float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
				float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

				// only care about the second intersection
				float tmax = t1;
				if( t2 > t1 )
					tmax = t2;

				// how far does the ball land horizontlly from the edge of the cliff?
				float upperDist = vx * tmax  -  g;

				// see if the ball hits the castle:
				if(  fabs( upperDist - d ) > TOL )
				{
					// if( DEBUG )  fprintf( stderr, "Misses the castle at upperDist = %8.3f\n", upperDist );
				}
				else
				{
					// if( DEBUG )  fprintf( stderr, "Hits the castle at upperDist = %8.3f\n", upperDist );
					numHits=1; //Add 1 to numHits edit
				}
			}
		} // if ball clears the cliff face
	} // if ball gets as far 

        dhits[gid] = numHits;
	
}
int main( int argc, char* argv[ ] )
{
        TimeOfDaySeed( );

        // int dev = findCudaDevice(argc, (const char **)argv);

        // better to define these here so that the rand() calls don't get into the thread timing:
        float *hvs   = new float [NUMTRIALS];
        float *hths  = new float [NUMTRIALS];
        float *hgs   = new float [NUMTRIALS];
        float *hhs   = new float [NUMTRIALS];
        float *hds   = new float [NUMTRIALS];
        int   *hhits = new int   [NUMTRIALS];

		// fill the random-value arrays:
		for( int n = 0; n < NUMTRIALS; n++ )
		{
			hvs[n]  = Ranf(  VMIN,  VMAX );
			hths[n] = Ranf( THMIN, THMAX );
			 hgs[n]  = Ranf(  GMIN,  GMAX );
			 hhs[n]  = Ranf(  HMIN,  HMAX );
			 hds[n]  = Ranf(  DMIN,  DMAX );
		}


        // allocate device memory:
        float *dvs, *dths, *dgs, *dhs, *dds;
        int   *dhits;

        hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
        hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
        CudaCheckError( );

        // copy host memory to the device:
        hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        CudaCheckError( );

        // setup the execution parameters:
        dim3 grid( NUMBLOCKS, 1, 1 );
        dim3 threads( BLOCKSIZE, 1, 1 );
        // allocate cuda events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop  );
        CudaCheckError( );

        // let the gpu go quiet:
        hipDeviceSynchronize( );

        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

        // execute the kernel:
        MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds,   OUT dhits );

        // record the stop event:
        hipEventRecord( stop, NULL );
        CudaCheckError( );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
        hipEventSynchronize( stop );
        CudaCheckError( );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, start, stop );
        CudaCheckError( );

        // compute and print the performance
		double secondsTotal = 0.001 * (double)msecTotal;
		double multsPerSecond = (double)NUMTRIALS / secondsTotal;
		double megaMultsPerSecond = multsPerSecond / 1000000.;
		// fprintf( stderr, "%12d\t%4d\t%10.2lf\n", NUMTRIALS, NUMBLOCKS, megaMultsPerSecond );

        // copy result from the device to the host:
        hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
        CudaCheckError( );

		long totalHits = 0;
		// int cliffFaceHits = 0;
		// int misses = 0;
		// int badError = 0;
		// int missCliff = 0;
        // add up the hhits[ ] array: :
		for(int i = 0; i < NUMTRIALS; i++){
			// if(hhits[i] > 0){
				totalHits+=hhits[i];
			// }
			// else if(hhits[i] == -10){
			// 	cliffFaceHits++;
			// }
			// else if(hhits[i] == -20){
			// 	misses++;
			// }
			// else if(hhits[i] == -100){
			// 	badError++;
			// }
			// else if(hhits[i] == 0){
			// 	missCliff++;
			// }
			// else{
			// 	printf("BAD %5d",hhits[i]);
			// }
		}
		// printf("%5d,%5d,%5d,%5d,%5d",missCliff,cliffFaceHits,misses,badError,totalHits);
		float hitPercentage = (float)totalHits*100.f/(float)NUMTRIALS;
		// printf("Hit Percentage = %10.2f%% \t",(float)totalHits*100.f/(float)NUMTRIALS);
		// printf("Total Hits = %d", totalHits);
        // compute and print the probability:
		fprintf( stderr, "%12d,%4d,%10.2lf,%10.2lf%%\n", NUMBLOCKS, BLOCKSIZE, megaMultsPerSecond, hitPercentage );

	// ?????

        // clean up host memory:
        delete [ ] hvs;
        delete [ ] hths;
        delete [ ] hgs;
        delete [ ] hhs;
        delete [ ] hds;
        delete [ ] hhits;

        // clean up device memory:
        hipFree( dvs );
        hipFree( dths );
        hipFree( dgs );
        hipFree( dhs );
        hipFree( dds );
        hipFree( dhits );
        CudaCheckError( );

	return 0;
}

